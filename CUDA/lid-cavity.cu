#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <>

#include "args.h"
#include "vtk.h"
#include "data.h"
#include "setup.h"

// Global timers
double total_time_build_rhs = 0.0;
double total_time_solve_poissons = 0.0;
double total_time_update_velocities = 0.0;
double total_time_apply_boundary = 0.0;

#define IDX(i,j) ((i)*(ny) + (j))

// Timer helper function
double get_time_seconds() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}


/**
 * @brief Build the right hand side of the Poisson equation on GPU.
 */
__global__ void kernel_build_rhs(double *u_d, double *v_d, double *b_d,
                                 double rho, double dt,
                                 double dx, double dy,
                                 int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
        double dudx = (u_d[IDX(i, j+1)] - u_d[IDX(i, j-1)]) / (2.0 * dx);
        double dvdy = (v_d[IDX(i+1, j)] - v_d[IDX(i-1, j)]) / (2.0 * dy);
        double dudx_sq = dudx * dudx;
        double dvdx = (v_d[IDX(i, j+1)] - v_d[IDX(i, j-1)]) / (2.0 * dx);
        double dudy = (u_d[IDX(i+1, j)] - u_d[IDX(i-1, j)]) / (2.0 * dy);
        double dvdy_sq = dvdy * dvdy;
        
        b_d[IDX(i,j)] = rho * (1.0 / dt) * (dudx + dvdy)
                        - dudx_sq - 2.0 * (dudy * dvdx) - dvdy_sq;
    }
}

/**
 * @brief Apply boundary conditions on GPU.
 */
__global__ void kernel_apply_boundary(double *u_d, double *v_d,
                                      int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // bottom row (j=0)
    if (i < nx) {
        u_d[IDX(i,0)] = 0.0;
        v_d[IDX(i,0)] = 0.0;
    }
    // top row (j=ny-1)
    if (i < nx && ny > 1) {
        u_d[IDX(i,ny-1)] = 0.0;
        v_d[IDX(i,ny-1)] = 0.0;
    }

    int j = blockIdx.y * blockDim.y + threadIdx.y;
    // left column (i=0)
    if (j < ny) {
        u_d[IDX(0,j)] = 0.0;
        v_d[IDX(0,j)] = 0.0;
    }
    // right column (i=nx-1)
    if (j < ny && nx > 1) {
        u_d[IDX(nx-1,j)] = 1.0; // // set the velocity on the cavity lid to 1.0
        v_d[IDX(nx-1,j)] = 0.0;
    }
}

/**
 * @brief One step of Poisson solver on GPU.
 */
__global__ void kernel_solve_poissons_step(double *p_d, double *pn_d, double *b_d,
                                           double dx, double dy,
                                           int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        // copy old data
        pn_d[IDX(i,j)] = p_d[IDX(i,j)];
    }

    __syncthreads();  

    if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
        p_d[IDX(i,j)] =
            (((pn_d[IDX(i,  j+1)] + pn_d[IDX(i,  j-1)]) * (dy*dy) +
              (pn_d[IDX(i+1,j)] + pn_d[IDX(i-1,j)]) * (dx*dx))
             / (2.0 * (dx*dx + dy*dy))
             - (dx*dx) * (dy*dy) / (2.0 * (dx*dx + dy*dy))
             * b_d[IDX(i,j)]);
    }
}

/**
 * @brief Apply poisson boundary conditions to p on GPU.
 */
__global__ void kernel_poisson_boundary(double *p_d, int nx, int ny)
{
    // for i in [0..nx-1], fix top/bottom
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nx) {
        p_d[IDX(i, ny-1)] = p_d[IDX(i, ny-2)];
        p_d[IDX(i, 0)]    = p_d[IDX(i, 1)];
    }

    // for j in [0..ny-1], fix left/right
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j < ny) {
        p_d[IDX(0, j)] = p_d[IDX(1, j)];
        p_d[IDX(nx-1, j)] = 0.0;
    }
}

/**
 * @brief Update velocities on GPU.
 */
__global__ void kernel_update_velocities(double *u_d, double *v_d,
                                         double *un_d, double *vn_d,
                                         double *p_d,
                                         double rho, double nu, double dt,
                                         double dx, double dy,
                                         int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        // store old velocities
        un_d[IDX(i,j)] = u_d[IDX(i,j)];
        vn_d[IDX(i,j)] = v_d[IDX(i,j)];
    }

    __syncthreads();

    if (i >= 1 && i < nx-1 && j >= 1 && j < ny-1) {
        // u-velocity
        double unij   = un_d[IDX(i,j)];
        double uni_jm = un_d[IDX(i,j-1)];
        double uni_mj = un_d[IDX(i-1,j)];
        double p_jp   = p_d[IDX(i,j+1)];
        double p_jm   = p_d[IDX(i,j-1)];

        u_d[IDX(i,j)] = unij
            - unij * dt / dx * (unij - uni_jm)
            - vn_d[IDX(i,j)] * dt / dy * (unij - uni_mj)
            - dt / (2.0 * rho * dx) * (p_jp - p_jm)
            + nu * (dt / (dx * dx) *
                    (un_d[IDX(i,j+1)] - 2.0 * unij + uni_jm)
                  + dt / (dy * dy) *
                    (un_d[IDX(i+1,j)] - 2.0 * unij + uni_mj));

        // v-velocity
        double vnij   = vn_d[IDX(i,j)];
        double vni_jm = vn_d[IDX(i,j-1)];
        double vni_mj = vn_d[IDX(i-1,j)];
        double p_ip   = p_d[IDX(i+1,j)];
        double p_im   = p_d[IDX(i-1,j)];

        v_d[IDX(i,j)] = vnij
            - un_d[IDX(i,j)] * dt / dx * (vnij - vni_jm)
            - vnij          * dt / dy * (vnij - vni_mj)
            - dt / (2.0 * rho * dy) * (p_ip - p_im)
            + nu * (dt / (dx * dx) *
                    (vn_d[IDX(i,j+1)] - 2.0 * vnij + vni_jm)
                  + dt / (dy * dy) *
                    (vn_d[IDX(i+1,j)] - 2.0 * vnij + vni_mj));
    }
}

/**
 * @brief Build the right hand side of the Poisson equation.
 */
void build_rhs() {
    double start_time = get_time_seconds();

    // Copy from host -> device
    hipMemcpy(d_u,  u[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v,  v[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(16,16);
    dim3 gridSize((nx+blockSize.x-1)/blockSize.x, (ny+blockSize.y-1)/blockSize.y);
    kernel_build_rhs<<<gridSize, blockSize>>>(d_u, d_v, d_b, rho, dt, dx, dy, nx, ny);

    hipDeviceSynchronize();

    // Copy back to host
    hipMemcpy(b[0], d_b, nx*ny*sizeof(double), hipMemcpyDeviceToHost);

    total_time_build_rhs += get_time_seconds() - start_time;
}
/**
 * @brief Apply the boundary conditions to the u and v arrays.
 */
void apply_boundary() {
    double start_time = get_time_seconds();

    // Copy current u, v to device
    hipMemcpy(d_u, u[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(16,16);
    dim3 gridSize((nx+blockSize.x-1)/blockSize.x, (ny+blockSize.y-1)/blockSize.y);
    kernel_apply_boundary<<<gridSize, blockSize>>>(d_u, d_v, nx, ny);
    
    hipDeviceSynchronize();

    // Copy back to host
    hipMemcpy(u[0], d_u, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(v[0], d_v, nx*ny*sizeof(double), hipMemcpyDeviceToHost);

    total_time_apply_boundary += get_time_seconds() - start_time;
}
/**
 * @brief Solve the Poisson equation to calculate the pressure across the domain.
 */
void solve_poissons() {
    double start_time = get_time_seconds();

    // Copy p, b to device
    hipMemcpy(d_p, p[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);

    //iteration on GPU
    dim3 blockSize(16,16);
    dim3 gridSize((nx+blockSize.x-1)/blockSize.x, (ny+blockSize.y-1)/blockSize.y);

    int poisson_iters = 0;
    while (poisson_iters < nit) {
        kernel_solve_poissons_step<<<gridSize, blockSize>>>(d_p, d_u, d_b, dx, dy, nx, ny);

        hipDeviceSynchronize();

        // apply boundary to p
        kernel_poisson_boundary<<<gridSize, blockSize>>>(d_p, nx, ny);

        hipDeviceSynchronize();

        poisson_iters++;
    }

    // Copy back to host
    hipMemcpy(p[0], d_p, nx*ny*sizeof(double), hipMemcpyDeviceToHost);

    total_time_solve_poissons += get_time_seconds() - start_time;
}
/**
 * @brief Update the u and v velocity arrays using the calculated pressure.
 */
void update_velocities() {
    double start_time = get_time_seconds();

    // Copy relevant arrays to device
    hipMemcpy(d_u,  u[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_v,  v[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p,  p[0], nx*ny*sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(16,16);
    dim3 gridSize((nx+blockSize.x-1)/blockSize.x, (ny+blockSize.y-1)/blockSize.y);
    kernel_update_velocities<<<gridSize, blockSize>>>(d_u, d_v, d_un, d_vn, d_p, rho, nu, dt, dx, dy, nx, ny);

    hipDeviceSynchronize();

    // Copy updated velocities back to host
    hipMemcpy(u[0], d_u, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(v[0], d_v, nx*ny*sizeof(double), hipMemcpyDeviceToHost);

    total_time_update_velocities += get_time_seconds() - start_time;
}
/**
 * @brief The main routine that sets up the problem and executes the timestepping routines
 * 
 * @param argc The number of arguments passed to the program
 * @param argv An array of the arguments passed to the program
 * @return int The return value of the application
 */
int main(int argc, char *argv[]) {
    double program_start_time = get_time_seconds();

    set_defaults();
    parse_args(argc, argv);
    setup();

    printf("Running problem size %f x %f on a %d x %d grid.\n", X, Y, nx, ny);

    if (verbose) print_opts();

    allocate_arrays();

    x = (double*)calloc(nx, sizeof(double));
    for (int i = 0; i < nx; i++) x[i] = (2.0 / (nx-1)) * i;
    y = (double*)calloc(ny, sizeof(double));
    for (int i = 0; i < ny; i++) y[i] = (2.0 / (ny-1)) * i;

    int iters = 0;
    double t = 0.0;

    while (iters < n_iters) {
        build_rhs();
        solve_poissons();
        update_velocities();
        apply_boundary();

        if (iters % output_freq == 0) {
            printf("Step %8d, Time: %14.8e (dt: %14.8e)\n", iters, t, dt);

            if ((!no_output) && (enable_checkpoints))
                write_checkpoint(iters);
        }

        iters++;
        t += dt;
    }

    printf("Step %8d, Time: %14.8e (dt: %14.8e)\n", iters, t, dt);
    printf("Simulation complete.\n");

    if (!no_output)
        write_result();

    free_arrays();

    double program_end_time = get_time_seconds();
    printf("\nExecution Summary:\n");
    printf("Total Program Time: %f seconds\n", program_end_time - program_start_time);
    printf("Total Time in build_rhs: %f seconds\n", total_time_build_rhs);
    printf("Total Time in solve_poissons: %f seconds\n", total_time_solve_poissons);
    printf("Total Time in update_velocities: %f seconds\n", total_time_update_velocities);
    printf("Total Time in apply_boundary: %f seconds\n", total_time_apply_boundary);

    exit(0);
}
